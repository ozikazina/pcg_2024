#include "hip/hip_runtime.h"
/**
 * @file      main.cu
 *
 * @author    Ondrej Vlcek \n
 *            Faculty of Information Technology \n
 *            Brno University of Technology \n
 *            xvlcek27@fit.vutbr.cz
 *
 * @brief     PCG Assignment 1
 *
 * @version   2024
 *
 * @date      04 October   2023, 09:00 (created) \n
 */

#include <cmath>
#include <cstdio>
#include <chrono>
#include <string>

#include "nbody.cuh"
#include "h5Helper.h"

/**
 * @brief CUDA error checking macro
 * @param call CUDA API call
 */
#define CUDA_CALL(call) \
  do { \
    const hipError_t _error = (call); \
    if (_error != hipSuccess) \
    { \
      std::fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, hipGetErrorString(_error)); \
      std::exit(EXIT_FAILURE); \
    } \
  } while(0)

/**
 * Main rotine
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv)
{
  if (argc != 10)
  {
    std::printf("Usage: nbody <N> <dt> <steps> <threads/block> <write intesity> <reduction threads> <reduction threads/block> <input> <output>\n");
    std::exit(1);
  }

  // Number of particles
  const unsigned N                   = static_cast<unsigned>(std::stoul(argv[1]));
  // Length of time step
  const float    dt                  = std::stof(argv[2]);
  // Number of steps
  const unsigned steps               = static_cast<unsigned>(std::stoul(argv[3]));
  // Number of thread blocks
  const unsigned simBlockDim         = static_cast<unsigned>(std::stoul(argv[4]));
  // Write frequency
  const unsigned writeFreq           = static_cast<unsigned>(std::stoul(argv[5]));
  // number of reduction threads
  const unsigned redTotalThreadCount = static_cast<unsigned>(std::stoul(argv[6]));
  // Number of reduction threads/blocks
  const unsigned redBlockDim         = static_cast<unsigned>(std::stoul(argv[7]));

  // Size of the simulation CUDA grid - number of blocks
  const unsigned simGridDim = (N + simBlockDim - 1) / simBlockDim;
  // Size of the reduction CUDA grid - number of blocks
  const unsigned redGridDim = (redTotalThreadCount + redBlockDim - 1) / redBlockDim;

  // Log benchmark setup
  std::printf("       NBODY GPU simulation\n"
              "N:                       %u\n"
              "dt:                      %f\n"
              "steps:                   %u\n"
              "threads/block:           %u\n"
              "blocks/grid:             %u\n"
              "reduction threads/block: %u\n"
              "reduction blocks/grid:   %u\n",
              N, dt, steps, simBlockDim, simGridDim, redBlockDim, redGridDim);

  const std::size_t recordsCount = (writeFreq > 0) ? (steps + writeFreq - 1) / writeFreq : 0;

  Particles hParticles{};

  hipHostAlloc(&hParticles.posX, N * sizeof(float), hipHostMallocMapped);
  hipHostAlloc(&hParticles.posY, N * sizeof(float), hipHostMallocMapped);
  hipHostAlloc(&hParticles.posZ, N * sizeof(float), hipHostMallocMapped);
  hipHostAlloc(&hParticles.velX, N * sizeof(float), hipHostMallocMapped);
  hipHostAlloc(&hParticles.velY, N * sizeof(float), hipHostMallocMapped);
  hipHostAlloc(&hParticles.velZ, N * sizeof(float), hipHostMallocMapped);
  hipHostAlloc(&hParticles.weight, N * sizeof(float), hipHostMallocMapped);

  MemDesc md(hParticles.posX,                 1,                          0,
             hParticles.posY,                 1,                          0,
             hParticles.posZ,                 1,                          0,
             hParticles.velX,                 1,                          0,
             hParticles.velY,                 1,                          0,
             hParticles.velZ,                 1,                          0,
             hParticles.weight,               1,                          0,
             N,
             recordsCount);

  // Initialisation of helper class and loading of input data
  H5Helper h5Helper(argv[8], argv[9], md);

  try
  {
    h5Helper.init();
    h5Helper.readParticleData();
  }
  catch (const std::exception& e)
  {
    std::fprintf(stderr, "Error: %s\n", e.what());
    return EXIT_FAILURE;
  }

  Particles  dParticles{};
  Velocities dTmpVelocities{};

  CUDA_CALL(hipMalloc(&dParticles.posX, N * sizeof(float)));
  CUDA_CALL(hipMalloc(&dParticles.posY, N * sizeof(float)));
  CUDA_CALL(hipMalloc(&dParticles.posZ, N * sizeof(float)));
  CUDA_CALL(hipMalloc(&dParticles.velX, N * sizeof(float)));
  CUDA_CALL(hipMalloc(&dParticles.velY, N * sizeof(float)));
  CUDA_CALL(hipMalloc(&dParticles.velZ, N * sizeof(float)));
  CUDA_CALL(hipMalloc(&dParticles.weight, N * sizeof(float)));

  CUDA_CALL(hipMalloc(&dTmpVelocities.x, N * sizeof(float)));
  CUDA_CALL(hipMalloc(&dTmpVelocities.y, N * sizeof(float)));
  CUDA_CALL(hipMalloc(&dTmpVelocities.z, N * sizeof(float)));

  CUDA_CALL(hipMemcpy(dParticles.posX, hParticles.posX, N * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dParticles.posY, hParticles.posY, N * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dParticles.posZ, hParticles.posZ, N * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dParticles.velX, hParticles.velX, N * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dParticles.velY, hParticles.velY, N * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dParticles.velZ, hParticles.velZ, N * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dParticles.weight, hParticles.weight, N * sizeof(float) , hipMemcpyHostToDevice));

  CUDA_CALL(hipMemset(dTmpVelocities.x, 0, N * sizeof(float)));
  CUDA_CALL(hipMemset(dTmpVelocities.y, 0, N * sizeof(float)));
  CUDA_CALL(hipMemset(dTmpVelocities.z, 0, N * sizeof(float)));

  // Start measurement
  const auto start = std::chrono::steady_clock::now();

  for (unsigned s = 0u; s < steps; ++s)
  {
    calculateGravitationVelocity<<<simGridDim, simBlockDim>>>(dParticles, dTmpVelocities, N, dt);
    calculateCollisionVelocity<<<simGridDim, simBlockDim>>>(dParticles, dTmpVelocities, N, dt);
    updateParticles<<<simGridDim, simBlockDim>>>(dParticles, dTmpVelocities, N, dt);
  }

  // Wait for all CUDA kernels to finish
  CUDA_CALL(hipDeviceSynchronize());

  // End measurement
  const auto end = std::chrono::steady_clock::now();

  // Approximate simulation wall time
  const float elapsedTime = std::chrono::duration<float>(end - start).count();
  std::printf("Time: %f s\n", elapsedTime);

  CUDA_CALL(hipMemcpy(hParticles.posX, dParticles.posX, N * sizeof(float), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.posY, dParticles.posY, N * sizeof(float), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.posZ, dParticles.posZ, N * sizeof(float), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.velX, dParticles.velX, N * sizeof(float), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.velY, dParticles.velY, N * sizeof(float), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.velZ, dParticles.velZ, N * sizeof(float), hipMemcpyDeviceToHost));

  // Compute reference center of mass on CPU
  const float4 refCenterOfMass = centerOfMassRef(md);

  std::printf("Reference center of mass: %f, %f, %f, %f\n",
              refCenterOfMass.x,
              refCenterOfMass.y,
              refCenterOfMass.z,
              refCenterOfMass.w);

  std::printf("Center of mass on GPU: %f, %f, %f, %f\n", 0.f, 0.f, 0.f, 0.f);

  // Writing final values to the file
  h5Helper.writeComFinal(refCenterOfMass);
  h5Helper.writeParticleDataFinal();

  CUDA_CALL(hipFree(dParticles.posX));
  CUDA_CALL(hipFree(dParticles.posY));
  CUDA_CALL(hipFree(dParticles.posZ));
  CUDA_CALL(hipFree(dParticles.velX));
  CUDA_CALL(hipFree(dParticles.velY));
  CUDA_CALL(hipFree(dParticles.velZ));
  CUDA_CALL(hipFree(dParticles.weight));

  CUDA_CALL(hipFree(dTmpVelocities.x));
  CUDA_CALL(hipFree(dTmpVelocities.y));
  CUDA_CALL(hipFree(dTmpVelocities.z));

  CUDA_CALL(hipHostFree(hParticles.posX));
  CUDA_CALL(hipHostFree(hParticles.posY));
  CUDA_CALL(hipHostFree(hParticles.posZ));
  CUDA_CALL(hipHostFree(hParticles.velX));
  CUDA_CALL(hipHostFree(hParticles.velY));
  CUDA_CALL(hipHostFree(hParticles.velZ));
  CUDA_CALL(hipHostFree(hParticles.weight));

}// end of main
//----------------------------------------------------------------------------------------------------------------------
